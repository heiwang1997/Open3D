#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------
// -                        Open3D: www.open3d.org                            -
// ----------------------------------------------------------------------------
// The MIT License (MIT)
//
// Copyright (c) 2018-2021 www.open3d.org
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
// FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
// IN THE SOFTWARE.
// ----------------------------------------------------------------------------

#include <hip/hip_runtime.h>

#include "open3d/core/CUDAUtils.h"
#include "open3d/core/ParallelFor.h"
#include "open3d/core/Tensor.h"
#include "open3d/t/pipelines/kernel/ComputeTransformImpl.h"
#include "open3d/t/pipelines/kernel/Reduction6x6Impl.cuh"
#include "open3d/t/pipelines/kernel/TransformationConverter.h"
#include "open3d/t/pipelines/registration/RobustKernel.h"
#include "open3d/t/pipelines/registration/RobustKernelImpl.h"

namespace open3d {
namespace t {
namespace pipelines {
namespace kernel {

const int kThread1DUnit = 256;

template <typename scalar_t, typename func_t>
__global__ void ComputePosePointToPlaneKernelCUDA(
        const scalar_t *source_points_ptr,
        const scalar_t *target_points_ptr,
        const scalar_t *target_normals_ptr,
        const int64_t *correspondence_indices,
        const int n,
        scalar_t *global_sum,
        func_t GetWeightFromRobustKernel) {
    __shared__ scalar_t local_sum0[kThread1DUnit];
    __shared__ scalar_t local_sum1[kThread1DUnit];
    __shared__ scalar_t local_sum2[kThread1DUnit];

    const int tid = threadIdx.x;

    local_sum0[tid] = 0;
    local_sum1[tid] = 0;
    local_sum2[tid] = 0;

    const int workload_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (workload_idx >= n) return;

    scalar_t J_ij[6] = {0}, reduction[29] = {0};
    scalar_t r = 0;

    bool valid = GetJacobianPointToPlane<scalar_t>(
            workload_idx, source_points_ptr, target_points_ptr,
            target_normals_ptr, correspondence_indices, J_ij, r);

    scalar_t w = GetWeightFromRobustKernel(r);

    if (valid) {
        // Dump J, r into JtJ and Jtr
        int i = 0;
        for (int j = 0; j < 6; ++j) {
            for (int k = 0; k <= j; ++k) {
                reduction[i] += J_ij[j] * w * J_ij[k];
                ++i;
            }
            reduction[21 + j] += J_ij[j] * w * r;
        }
        reduction[27] += r;
        reduction[28] += 1;
    }

    ReduceSum6x6LinearSystem<scalar_t, kThread1DUnit>(tid, valid, reduction,
                                                      local_sum0, local_sum1,
                                                      local_sum2, global_sum);
}

void ComputePosePointToPlaneCUDA(const core::Tensor &source_points,
                                 const core::Tensor &target_points,
                                 const core::Tensor &target_normals,
                                 const core::Tensor &correspondence_indices,
                                 core::Tensor &pose,
                                 float &residual,
                                 int &inlier_count,
                                 const core::Dtype &dtype,
                                 const core::Device &device,
                                 const registration::RobustKernel &kernel) {
    int n = source_points.GetLength();

    core::Tensor global_sum = core::Tensor::Zeros({29}, dtype, device);
    const dim3 blocks((n + kThread1DUnit - 1) / kThread1DUnit);
    const dim3 threads(kThread1DUnit);

    DISPATCH_FLOAT_DTYPE_TO_TEMPLATE(dtype, [&]() {
        scalar_t *global_sum_ptr = global_sum.GetDataPtr<scalar_t>();

        DISPATCH_ROBUST_KERNEL_FUNCTION(
                kernel.type_, scalar_t, kernel.scaling_parameter_,
                kernel.shape_parameter_, [&]() {
                    ComputePosePointToPlaneKernelCUDA<<<
                            blocks, threads, 0, core::cuda::GetStream()>>>(
                            source_points.GetDataPtr<scalar_t>(),
                            target_points.GetDataPtr<scalar_t>(),
                            target_normals.GetDataPtr<scalar_t>(),
                            correspondence_indices.GetDataPtr<int64_t>(), n,
                            global_sum_ptr, GetWeightFromRobustKernel);
                });
    });

    core::cuda::Synchronize();

    DecodeAndSolve6x6(global_sum, pose, residual, inlier_count);
}

}  // namespace kernel
}  // namespace pipelines
}  // namespace t
}  // namespace open3d
